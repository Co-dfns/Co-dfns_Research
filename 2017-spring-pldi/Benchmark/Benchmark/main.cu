#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <ctime>
#include <iostream>
#include <string>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>

using namespace std;

struct CPUNode {
	long long id;
	short type;
	vector<CPUNode> kids;
	CPUNode();
	CPUNode(int depth, int width, int& id);
	CPUNode(long long id, short type, vector<CPUNode> kids);
	friend ostream& operator<<(ostream& os, const CPUNode& node);
};

CPUNode::CPUNode()
	: id(0),type(1),kids(vector<CPUNode>())
{}

CPUNode::CPUNode(int depth, int width, int& id)
	: id(id++)
{
	if (depth) {
		type = 0;
		kids = vector<CPUNode>(width);
		for (CPUNode& node : kids)
			node = CPUNode(depth - 1, width, id);
	} else {
		type = 1;
		kids = vector<CPUNode>();
	}
}

CPUNode::CPUNode(long long id, short type, vector<CPUNode> kids)
	: id(id),type(type),kids(kids)
{}

ostream& operator<<(ostream& os, const CPUNode& node)
{
	os << node.type << " " << node.id;
	return os;
}

void print_cpu_tree(CPUNode& node, int depth)
{
	for (int i = 0; i < depth; i++)
		cout << " ";

	cout << node << endl;

	for (CPUNode& kid : node.kids)
		print_cpu_tree(kid, depth + 1);
}

void help(void) 
{
	cout << "Benchmark <cpu|gpu> <print|quiet> [<depth> <width>]" << endl;
}

void cpu_flatten_helper(CPUNode& node, vector<CPUNode>& lifted)
{
	if (!node.type) {
		for_each(node.kids.rbegin(), node.kids.rend(), [&lifted](CPUNode& kid) {
			cpu_flatten_helper(kid, lifted);
		});

		lifted.push_back(node);
		node.type = 1;
		node.kids = vector<CPUNode>();
	}
}

void cpu_flatten(CPUNode& node)
{
	vector<CPUNode> lifted;

	cpu_flatten_helper(node, lifted);

	node = CPUNode{ -1, 0, lifted };
}

void benchmark_cpu(int depth, int width, bool print)
{
	cout << "Benchmarking CPU algorithm (Depth: " << depth << " Width: " << width << ")" << endl;
	cout << "Creating AST...";

	int id = 0;
	CPUNode node(depth, width, id);

	cout << "done." << endl;

	if (print) {
		cout << endl << "Before: " << endl;
		print_cpu_tree(node, 1);
		cout << endl;
	}

	cout << "Flattening AST...";

	auto start = chrono::high_resolution_clock::now();
	cpu_flatten(node);
	auto end = chrono::high_resolution_clock::now();

	cout << "took " << chrono::duration_cast<chrono::milliseconds>(end - start).count()
		<< " milliseconds." << endl;

	if (print) {
		cout << endl << "After: " << endl;
		print_cpu_tree(node, 1);
	}
}

struct print_gpu_node {
	template <typename Tuple>
	__host__
	void operator()(Tuple t) 
	{
		for (int i = 0; i <= thrust::get<1>(t); i++)
			cout << " ";

		cout << thrust::get<2>(t) << " " << thrust::get<0>(t) << " ";

		for (int v : thrust::get<3>(t))
			cout << " " << v;

		cout << endl;
	}
};

void print_gpu_tree(thrust::host_vector<int> depths,
	thrust::host_vector<long long> ids,
	thrust::host_vector<short> types,
	thrust::host_vector<vector<int>> coords)
{
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(ids.begin(), depths.begin(), types.begin(), coords.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(ids.end(), depths.end(), types.end(), coords.end())),
		print_gpu_node());
}

void benchmark_gpu(int depth, int width, bool print)
{
	cout << "Benchmarking GPU algorithm (Depth: " << depth << " Width: " << width << ")..." << endl;
	cout << "Creating AST...";

	long long count = 0;

	for (int i = 0; i <= depth; i++) {
		count += pow(width, i);
	}

	thrust::host_vector<int> depths(count);
	thrust::host_vector<long long> ids(count);
	thrust::host_vector<short> types(count);
	thrust::host_vector<vector<int>> coords(count);

	int cur_depth = 0;
	int cur_width = 0;
	vector<int> cur_coord(depth+1, 0);

	for (int i = 0; i < count; i++) {
		if (cur_width >= width) {
			cur_coord[cur_depth] = 0;
			cur_depth--;
			cur_width = cur_coord[cur_depth];
			i--;
			continue;
		}

		depths[i] = cur_depth;
		types[i] = cur_depth == depth ? 1 : 0;
		cur_coord[cur_depth]++;
		coords[i] = cur_coord;
		cur_width++;

		if (cur_depth < depth) {
			cur_depth++;
			cur_width = 0;
		}
	}

	thrust::sequence(ids.begin(), ids.end());

	cout << "done." << endl;

	if (print) {
		cout << endl << "Before: " << endl;
		print_gpu_tree(depths, ids, types, coords);
		cout << endl;
	}

	cout << "Flattening AST...";

	auto start = chrono::high_resolution_clock::now();
	
	auto end = chrono::high_resolution_clock::now();

	cout << "took " << chrono::duration_cast<chrono::milliseconds>(end - start).count()
		<< " milliseconds." << endl;

	if (print) {
		cout << endl << "After: " << endl;
		print_gpu_tree(depths, ids, types, coords);
	}
}

int main(int argc, char *argv[])
{
	int depth = 3;
	int width = 2;
	bool print = true;

	string print_str("print");
	string quiet_str("quiet");
	string cpu_str("cpu");
	string gpu_str("gpu");

	if (argc != 3 && argc != 5) {
		help();
		return 1;
	}

	if (argc == 5) {
		depth = stoi(argv[3]);
		width = stoi(argv[4]);
	}

	if (print_str == argv[2])
		print = true;
	else if (quiet_str == argv[2])
		print = false;
	else {
		help();
		return 1;
	}

	if (cpu_str == argv[1])
		benchmark_cpu(depth, width, print);
	else if (gpu_str == argv[1])
		benchmark_gpu(depth, width, print);
	else {
		help();
		return 1;
	}

	return 0;
}