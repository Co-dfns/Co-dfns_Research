#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <ctime>
#include <iostream>
#include <string>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/tuple.h>

using namespace std;

struct CPUNode {
	long long id;
	short type;
	vector<CPUNode> kids;
	CPUNode();
	CPUNode(int depth, int width, int& id);
	CPUNode(long long id, short type, vector<CPUNode> kids);
	friend ostream& operator<<(ostream& os, const CPUNode& node);
};

CPUNode::CPUNode()
	: id(0),type(1),kids(vector<CPUNode>())
{}

CPUNode::CPUNode(int depth, int width, int& id)
	: id(id++)
{
	if (depth) {
		type = 0;
		kids = vector<CPUNode>(width);
		for (CPUNode& node : kids)
			node = CPUNode(depth - 1, width, id);
	} else {
		type = 1;
		kids = vector<CPUNode>();
	}
}

CPUNode::CPUNode(long long id, short type, vector<CPUNode> kids)
	: id(id),type(type),kids(kids)
{}

ostream& operator<<(ostream& os, const CPUNode& node)
{
	os << node.type << " " << node.id;
	return os;
}

void print_cpu_tree(CPUNode& node, int depth)
{
	for (int i = 0; i < depth; i++)
		cout << " ";

	cout << node << endl;

	for (CPUNode& kid : node.kids)
		print_cpu_tree(kid, depth + 1);
}

void help(void) 
{
	cout << "Benchmark <cpu|gpu> <print|quiet> [<depth> <width>]" << endl;
}

void cpu_flatten_helper(CPUNode& node, vector<CPUNode>& lifted)
{
	if (!node.type) {
		for_each(node.kids.rbegin(), node.kids.rend(), [&lifted](CPUNode& kid) {
			cpu_flatten_helper(kid, lifted);
		});

		lifted.push_back(node);
		node.type = 1;
		node.kids = vector<CPUNode>();
	}
}

void cpu_flatten(CPUNode& node)
{
	vector<CPUNode> lifted;

	cpu_flatten_helper(node, lifted);

	node = CPUNode{ -1, 0, lifted };
}

void benchmark_cpu(int depth, int width, bool print)
{
	cout << "Benchmarking CPU algorithm (Depth: " << depth << " Width: " << width << ")" << endl;
	cout << "Creating AST...";

	int id = 0;
	CPUNode node(depth, width, id);

	cout << "done." << endl;

	if (print) {
		cout << endl << "Before: " << endl;
		print_cpu_tree(node, 1);
		cout << endl;
	}

	cout << "Flattening AST...";

	auto start = chrono::high_resolution_clock::now();
	cpu_flatten(node);
	auto end = chrono::high_resolution_clock::now();

	cout << "took " << chrono::duration_cast<chrono::milliseconds>(end - start).count()
		<< " milliseconds." << endl;

	if (print) {
		cout << endl << "After: " << endl;
		print_cpu_tree(node, 1);
	}
}

struct GPUNode {
	int depth;
	int width;
	long long count;
	thrust::device_vector<int> depths;
	thrust::device_vector<short> types;
	thrust::device_vector<int> coords;
	GPUNode(int depth, int width);
};

GPUNode::GPUNode(int depth, int width)
	: depth(depth), width(width)
{
	count = 0;

	for (int i = 0; i < depth; i++)
		count += (long long)pow(width, i);

	thrust::host_vector<int> host_depths(count);
	thrust::host_vector<short> host_types(count);
	thrust::host_vector<int> host_coords(count * depth, 0);

	int cur_width = 0;
	vector<int> cur_coord(1, 0);

	for (int i = 0; i < count; i++) {
		if (cur_width >= width) {
			cur_coord.pop_back();
			cur_width = cur_coord.back();
			i--;
			continue;
		}

		host_depths[i] = (int)cur_coord.size() - 1;
		host_types[i] = cur_coord.size() >= depth ? 1 : 0;
		cur_coord.back()++;
		for (int j = 0; j < cur_coord.size(); j++)
			host_coords[i*depth + j] = cur_coord[j];
		cur_width++;

		if (cur_coord.size() < depth) {
			cur_width = 0;
			cur_coord.push_back(0);
		}
	}

	depths = host_depths;
	types = host_types;
	coords = host_coords;
}

struct print_gpu_node {
	int max_depth;
	thrust::host_vector<int>& coords;

	template <typename Tuple>
	__host__
	void operator()(Tuple t) 
	{
		int depth = thrust::get<0>(t) + 1;

		for (int i = 0; i < depth; i++)
			cout << " ";

		cout << thrust::get<1>(t) << " ";

		long long i = thrust::get<2>(t);

		for (int j = 0; j < max_depth; j++) {
			int c = coords[i*max_depth + j];
			if (c)
				cout << " " << c;
			else
				break;
		}

		cout << endl;
	}
};

void print_gpu_tree(GPUNode& ast)
{
	thrust::host_vector<int> host_depths = ast.depths;
	thrust::host_vector<short> host_types = ast.types;
	thrust::host_vector<int> host_coords = ast.coords;
	thrust::counting_iterator<long long> row(0);

	thrust::for_each(thrust::host,
		thrust::make_zip_iterator(
			thrust::make_tuple(host_depths.begin(), host_types.begin(), row)),
		thrust::make_zip_iterator(
			thrust::make_tuple(host_depths.end(), host_types.end(), row + ast.count)),
		print_gpu_node{ ast.depth, host_coords});
}

typedef thrust::tuple<long long, long long> cpitype;

struct coord_parent_index {
	int max_depth;
	long long exp_count;
	thrust::device_ptr<int> coords;
	thrust::device_ptr<long long> eids;

	coord_parent_index(int md, long long ec, thrust::device_ptr<int> cs, thrust::device_ptr<long long> eids)
		: max_depth(md), exp_count(ec), coords(cs), eids(eids)
	{}

	__host__ __device__
	bool test(const cpitype& t)
	{
		long long ci = thrust::get<0>(t);
		long long ei = thrust::get<1>(t);

		for (int j = 0; j < max_depth - 1; j++) {
			int ref = coords[ei*max_depth + j];
			int cor = coords[ci*max_depth + j];
			int nxt = coords[ci*max_depth + j + 1];

			if (!nxt) {
				if (ref) return false;
				else break;
			}
			if (!ref) break;
			if (ref != cor) return false;
		}

		return true;
	}

	__host__ __device__
	cpitype operator()(const cpitype& t1, const cpitype& t2)
	{
		long long v1 = thrust::get<1>(t1);
		long long v2 = thrust::get<1>(t2);

		if (v1 >= v2) {
			if (test(t1))
				return t1;
			else if (test(t2))
				return t2;
			else
				return thrust::make_tuple<long long, long long>(0, 0);
		}
		else {
			if (test(t2))
				return t2;
			else if (test(t1))
				return t1;
			else
				return thrust::make_tuple<long long, long long>(0, 0);
		}
	}
};

struct copy_coord {
	thrust::device_ptr<int> new_coords;
	thrust::device_ptr<int> old_coords;
	int max_depth;

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		long long ci = thrust::get<0>(t);
		long long ei = thrust::get<1>(t);
		for (int j = 0; j < max_depth; j++)
			new_coords[ci*max_depth + j] = old_coords[ei*max_depth + j];
	}
};

struct get_ci {
	long long C;

	__host__ __device__
	long long operator()(long long i)
	{
		return 1 + i / C;
	}
};

struct get_ei {
	long long C;
	thrust::device_ptr<long long> eids;

	__host__ __device__
	long long operator()(long long i)
	{
		return eids[i % C];
	}
};

void gpu_flatten(GPUNode& ast)
{
	thrust::device_vector<long long> eids(ast.count);
	thrust::sequence(eids.begin(), eids.end());

	auto eids_begin = eids.begin();
	auto eids_end = thrust::remove_if(eids_begin, eids.end(), ast.types.begin(), 
		thrust::identity<short>());

	long long exp_count = eids_end - eids_begin;
	long long result_count = ast.count + exp_count - 1;

	thrust::device_vector<int> new_depths(result_count);
	thrust::device_vector<short> new_types(result_count);
	thrust::device_vector<int> new_coords(result_count * ast.depth);
	thrust::device_vector<long long> refids(result_count);
	thrust::device_vector<long long> keys(result_count);
	thrust::counting_iterator<long long> cids_begin(1);
	thrust::counting_iterator<long long> newids(0);

	thrust::fill(new_depths.begin(), new_depths.begin() + exp_count, 0);
	thrust::fill(new_depths.begin() + exp_count, new_depths.end(), 1);
	thrust::fill(new_types.begin(), new_types.begin() + exp_count, 0);
	thrust::fill(new_types.begin() + exp_count, new_types.end(), 1);

	auto keys_first = keys.begin() + exp_count;

	thrust::copy(eids_begin, eids_end, keys.begin());
	thrust::reduce_by_key(
		thrust::make_transform_iterator(newids, get_ci{ exp_count }),
		thrust::make_transform_iterator(newids + exp_count * (ast.count - 1), get_ci{ exp_count }),
		thrust::make_zip_iterator(thrust::make_tuple(
			thrust::make_transform_iterator(newids, get_ci{ exp_count }),
			thrust::make_transform_iterator(newids, get_ei{ exp_count, eids.data() }))),
		thrust::make_discard_iterator(),
		thrust::make_zip_iterator(thrust::make_tuple(refids.begin(), keys_first)),
		thrust::equal_to<long long>(),
		coord_parent_index(ast.depth, exp_count, ast.coords.data(), eids.data()));
	thrust::copy(eids_begin, eids_end, refids.begin());
	thrust::sequence(refids.begin() + exp_count, refids.end(), 1);
	thrust::stable_sort_by_key(keys.begin(), keys.end(),
		thrust::make_zip_iterator(
			thrust::make_tuple(new_depths.begin(), new_types.begin(), refids.begin())));
	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(newids, refids.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(newids + result_count, refids.end())),
		copy_coord{ new_coords.data(), ast.coords.data(), ast.depth });

	ast.count = result_count;
	ast.types = new_types;
	ast.coords = new_coords;
	ast.depths = new_depths;
}

void benchmark_gpu(int depth, int width, bool print)
{
	cout << "Benchmarking GPU algorithm (Depth: " << depth << " Width: " << width << ")..." << endl;
	cout << "Creating AST...";

	GPUNode ast(depth+1, width);

	cout << "done." << endl;

	if (print) {
		cout << endl << "Before: " << endl;
		print_gpu_tree(ast);
		cout << endl;
	}

	cout << "Flattening AST...";

	auto start = chrono::high_resolution_clock::now();
	gpu_flatten(ast);
	auto end = chrono::high_resolution_clock::now();

	cout << "took " << chrono::duration_cast<chrono::milliseconds>(end - start).count()
		<< " milliseconds." << endl;

	if (print) {
		cout << endl << "After: " << endl;
		print_gpu_tree(ast);
	}
}

int main(int argc, char *argv[])
{
	int depth = 3;
	int width = 2;
	bool print = true;

	string print_str("print");
	string quiet_str("quiet");
	string cpu_str("cpu");
	string gpu_str("gpu");

	if (argc != 3 && argc != 5) {
		help();
		return 1;
	}

	if (argc == 5) {
		depth = stoi(argv[3]);
		width = stoi(argv[4]);
	}

	if (print_str == argv[2])
		print = true;
	else if (quiet_str == argv[2])
		print = false;
	else {
		help();
		return 1;
	}

	if (cpu_str == argv[1])
		benchmark_cpu(depth, width, print);
	else if (gpu_str == argv[1])
		benchmark_gpu(depth, width, print);
	else {
		help();
		return 1;
	}

	return 0;
}