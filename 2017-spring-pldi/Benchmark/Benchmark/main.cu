#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <ctime>
#include <iostream>
#include <string>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/sort.h>

using namespace std;

struct CPUNode {
	long long id;
	short type;
	vector<CPUNode> kids;
	CPUNode();
	CPUNode(int depth, int width, int& id);
	CPUNode(long long id, short type, vector<CPUNode> kids);
	friend ostream& operator<<(ostream& os, const CPUNode& node);
};

CPUNode::CPUNode()
	: id(0),type(1),kids(vector<CPUNode>())
{}

CPUNode::CPUNode(int depth, int width, int& id)
	: id(id++)
{
	if (depth) {
		type = 0;
		kids = vector<CPUNode>(width);
		for (CPUNode& node : kids)
			node = CPUNode(depth - 1, width, id);
	} else {
		type = 1;
		kids = vector<CPUNode>();
	}
}

CPUNode::CPUNode(long long id, short type, vector<CPUNode> kids)
	: id(id),type(type),kids(kids)
{}

ostream& operator<<(ostream& os, const CPUNode& node)
{
	os << node.type << " " << node.id;
	return os;
}

void print_cpu_tree(CPUNode& node, int depth)
{
	for (int i = 0; i < depth; i++)
		cout << " ";

	cout << node << endl;

	for (CPUNode& kid : node.kids)
		print_cpu_tree(kid, depth + 1);
}

void help(void) 
{
	cout << "Benchmark <cpu|gpu> <print|quiet> [<depth> <width>]" << endl;
}

void cpu_flatten_helper(CPUNode& node, vector<CPUNode>& lifted)
{
	if (!node.type) {
		for_each(node.kids.rbegin(), node.kids.rend(), [&lifted](CPUNode& kid) {
			cpu_flatten_helper(kid, lifted);
		});

		lifted.push_back(node);
		node.type = 1;
		node.kids = vector<CPUNode>();
	}
}

void cpu_flatten(CPUNode& node)
{
	vector<CPUNode> lifted;

	cpu_flatten_helper(node, lifted);

	node = CPUNode{ -1, 0, lifted };
}

void benchmark_cpu(int depth, int width, bool print)
{
	cout << "Benchmarking CPU algorithm (Depth: " << depth << " Width: " << width << ")" << endl;
	cout << "Creating AST...";

	int id = 0;
	CPUNode node(depth, width, id);

	cout << "done." << endl;

	if (print) {
		cout << endl << "Before: " << endl;
		print_cpu_tree(node, 1);
		cout << endl;
	}

	cout << "Flattening AST...";

	auto start = chrono::high_resolution_clock::now();
	cpu_flatten(node);
	auto end = chrono::high_resolution_clock::now();

	cout << "took " << chrono::duration_cast<chrono::milliseconds>(end - start).count()
		<< " milliseconds." << endl;

	if (print) {
		cout << endl << "After: " << endl;
		print_cpu_tree(node, 1);
	}
}

struct GPUNode {
	int depth;
	int width;
	int count;
	thrust::host_vector<int> depths;
	thrust::host_vector<long long> ids;
	thrust::host_vector<short> types;
	thrust::host_vector<vector<int>> coords;
	GPUNode(int depth, int width);
};

GPUNode::GPUNode(int depth, int width)
	: depth(depth), width(width)
{
	count = 0;

	for (int i = 0; i <= depth; i++)
		count += pow(width, i);

	depths = thrust::host_vector<int>(count);
	ids = thrust::host_vector<long long>(count);
	types = thrust::host_vector<short>(count);
	coords = thrust::host_vector<vector<int>>(count);

	int cur_width = 0;
	vector<int> cur_coord(1, 0);

	for (int i = 0; i < count; i++) {
		if (cur_width >= width) {
			cur_coord.pop_back();
			cur_width = cur_coord.back();
			i--;
			continue;
		}

		depths[i] = cur_coord.size() - 1;
		types[i] = cur_coord.size() > depth ? 1 : 0;
		cur_coord.back()++;
		coords[i] = cur_coord;
		cur_width++;

		if (cur_coord.size() <= depth) {
			cur_width = 0;
			cur_coord.push_back(0);
		}
	}

	thrust::sequence(ids.begin(), ids.end());
}

struct print_gpu_node {
	template <typename Tuple>
	__host__
	void operator()(Tuple t) 
	{
		for (int i = 0; i <= thrust::get<1>(t); i++)
			cout << " ";

		cout << thrust::get<2>(t) << " " << thrust::get<0>(t) << " ";

		for (int v : thrust::get<3>(t))
			cout << " " << v;

		cout << endl;
	}
};

void print_gpu_tree(GPUNode& ast)
{
	thrust::for_each(
		thrust::make_zip_iterator(
			thrust::make_tuple(ast.ids.begin(), ast.depths.begin(), ast.types.begin(), ast.coords.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(ast.ids.end(), ast.depths.end(), ast.types.end(), ast.coords.end())),
		print_gpu_node());
}

struct print_gpu_reference {
	__host__
	void operator()(vector<int>& ref)
	{
		for (int& v : ref)
			cout << v << " ";
		cout << endl;
	}
};

struct coord_prefix_index {
	template <typename Tuple>
	__host__
	long long operator()(Tuple t)
	{
		vector<int>& eref = thrust::get<1>(t);
		vector<int>& coord = thrust::get<2>(t);

		if (eref.size() >= coord.size())
			return 0;

		for (int i = 0; i < eref.size(); i++)
			if (eref[i] != coord[i])
				return 0;

		return thrust::get<0>(t);
	}
};

struct coord_parent_index {
	thrust::detail::normal_iterator<vector<int>*> first;
	thrust::detail::normal_iterator<vector<int>*> last;
	__host__
	long long operator()(vector<int>& ref)
	{
		long long ecount = last - first;
		thrust::counting_iterator<long long> count(0);
		thrust::constant_iterator<vector<int>&> coord(ref);

		return thrust::transform_reduce(
			thrust::make_zip_iterator(thrust::make_tuple(count, first, coord)),
			thrust::make_zip_iterator(thrust::make_tuple(count + ecount, last, coord + ecount)),
			coord_prefix_index(), 0, thrust::maximum<int>());
	}
};

void gpu_flatten(GPUNode& ast)
{
	thrust::host_vector<vector<int>> ecs(ast.count);
	auto ecs_end = 
		thrust::remove_copy_if(
			ast.coords.begin(), ast.coords.end(), ast.types.begin(), ecs.begin(), 
			thrust::identity<int>());

	thrust::host_vector<long long> keys(ast.count);
	thrust::transform(ast.coords.begin(), ast.coords.end(), keys.begin(), coord_parent_index{ ecs.begin(), ecs_end });
	thrust::host_vector<long long> perm(ast.count);
	thrust::sequence(perm.begin(), perm.end());
	thrust::stable_sort_by_key(keys.begin(), keys.end(), perm.begin());

	cout << endl;
	auto ast_tuple = thrust::make_zip_iterator(
		thrust::make_tuple(
			ast.ids.begin(), ast.depths.begin(), ast.types.begin(), ast.coords.begin()));
	thrust::for_each(
		thrust::make_permutation_iterator(ast_tuple, perm.begin()),
		thrust::make_permutation_iterator(ast_tuple, perm.end()),
		print_gpu_node());
	cout << endl;

	// Sort nodes
	// Scatter Nodes?
	// Adjust depths, set types, set ref
}

void benchmark_gpu(int depth, int width, bool print)
{
	cout << "Benchmarking GPU algorithm (Depth: " << depth << " Width: " << width << ")..." << endl;
	cout << "Creating AST...";

	GPUNode ast(depth, width);

	cout << "done." << endl;

	if (print) {
		cout << endl << "Before: " << endl;
		print_gpu_tree(ast);
		cout << endl;
	}

	cout << "Flattening AST...";

	auto start = chrono::high_resolution_clock::now();
	gpu_flatten(ast);
	auto end = chrono::high_resolution_clock::now();

	cout << "took " << chrono::duration_cast<chrono::milliseconds>(end - start).count()
		<< " milliseconds." << endl;

	if (print) {
		cout << endl << "After: " << endl;
		print_gpu_tree(ast);
	}
}

int main(int argc, char *argv[])
{
	int depth = 3;
	int width = 2;
	bool print = true;

	string print_str("print");
	string quiet_str("quiet");
	string cpu_str("cpu");
	string gpu_str("gpu");

	if (argc != 3 && argc != 5) {
		help();
		return 1;
	}

	if (argc == 5) {
		depth = stoi(argv[3]);
		width = stoi(argv[4]);
	}

	if (print_str == argv[2])
		print = true;
	else if (quiet_str == argv[2])
		print = false;
	else {
		help();
		return 1;
	}

	if (cpu_str == argv[1])
		benchmark_cpu(depth, width, print);
	else if (gpu_str == argv[1])
		benchmark_gpu(depth, width, print);
	else {
		help();
		return 1;
	}

	return 0;
}