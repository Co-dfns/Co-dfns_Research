#include "hip/hip_runtime.h"
﻿#include <algorithm>
#include <chrono>
#include <cmath>
#include <ctime>
#include <iostream>
#include <string>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/tuple.h>
#include <thrust/equal.h>
#include <thrust/transform.h>

using namespace std;

struct CPUNode {
	long long id;
	short type;
	vector<CPUNode> kids;
	CPUNode();
	CPUNode(int depth, int width, int& id);
	CPUNode(long long id, short type, vector<CPUNode> kids);
	friend ostream& operator<<(ostream& os, const CPUNode& node);
};

CPUNode::CPUNode()
	: id(0),type(1),kids(vector<CPUNode>())
{}

CPUNode::CPUNode(int depth, int width, int& id)
	: id(id++)
{
	if (depth) {
		type = 0;
		kids = vector<CPUNode>(width);
		for (CPUNode& node : kids)
			node = CPUNode(depth - 1, width, id);
	} else {
		type = 1;
		kids = vector<CPUNode>();
	}
}

CPUNode::CPUNode(long long id, short type, vector<CPUNode> kids)
	: id(id),type(type),kids(kids)
{}

ostream& operator<<(ostream& os, const CPUNode& node)
{
	os << node.type << " " << node.id;
	return os;
}

void print_cpu_tree(CPUNode& node, int depth)
{
	for (int i = 0; i < depth; i++)
		cout << " ";

	cout << node << endl;

	for (CPUNode& kid : node.kids)
		print_cpu_tree(kid, depth + 1);
}

void help(void) 
{
	cout << "Benchmark <fnc|mut|gpu> <print|quiet> [<depth> <width>|<depth_start> <depth_end> <width_start> <width_end>]" << endl;
}

CPUNode functional_flatten(CPUNode& node)
{
	vector<CPUNode> nodes;

	if (!node.type) {
		vector<CPUNode> kids(node.kids.size());

		for (int i = 0; i < node.kids.size(); i++) {
			kids[i].id = node.kids[i].id;
			kids[i].type = 1;
			kids[i].kids = vector<CPUNode>();
		}

		nodes.push_back(CPUNode(node.id, node.type, kids));

		for (int i = 0; i < node.kids.size(); i++) {
			auto f = functional_flatten(node.kids[i]);
			for (auto k : f.kids) nodes.push_back(k);
		}
	}

	return CPUNode(-1, 0, nodes);
}

void benchmark_functional(int depth, int width, bool print)
{
	cout << "Benchmarking Functional algorithm (Depth: " << depth << " Width: " << width << ")" << endl;
	cout << "Creating AST...";

	int id = 0;
	CPUNode node(depth, width, id);

	cout << "done." << endl;

	if (print) {
		cout << endl << "Before: " << endl;
		print_cpu_tree(node, 1);
		cout << endl;
	}

	cout << "Flattening AST...";

	long long timing = 0;

	CPUNode temp;

	for (int i = 0; i < 5; i++) {
		auto start = chrono::high_resolution_clock::now();
		temp = functional_flatten(node);
		auto end = chrono::high_resolution_clock::now();
		timing += chrono::duration_cast<chrono::microseconds>(end - start).count();
	}

	node = temp;

	double average_timing = (double)timing / 5;

	cout << "took an average of " << average_timing / 1000 << " milliseconds." << endl;
	cout << "SET_TIMINGS 0 " << depth << " " << width << " " << average_timing / 1000 << endl;

	if (print) {
		cout << endl << "After: " << endl;
		print_cpu_tree(node, 1);
	}
}

void mutation_flatten_helper(CPUNode& node, vector<CPUNode>& lifted)
{
	if (!node.type) {
		for_each(node.kids.rbegin(), node.kids.rend(), [&lifted](CPUNode& kid) {
			mutation_flatten_helper(kid, lifted);
		});

		lifted.push_back(node);
		node.type = 1;
		node.kids = vector<CPUNode>();
	}
}

void mutation_flatten(CPUNode& node)
{
	vector<CPUNode> lifted;

	mutation_flatten_helper(node, lifted);

	node = CPUNode{ -1, 0, lifted };
}

void benchmark_mutation(int depth, int width, bool print)
{
	cout << "Benchmarking Mutation algorithm (Depth: " << depth << " Width: " << width << ")" << endl;
	cout << "Creating AST...";

	int id = 0;
	CPUNode node(depth, width, id);

	cout << "done." << endl;

	if (print) {
		cout << endl << "Before: " << endl;
		print_cpu_tree(node, 1);
		cout << endl;
	}

	cout << "Flattening AST...";

	long long timing = 0;

	CPUNode temp;

	for (int i = 0; i < 5; i++) {
		temp = node;
		auto start = chrono::high_resolution_clock::now();
		mutation_flatten(temp);
		auto end = chrono::high_resolution_clock::now();
		timing += chrono::duration_cast<chrono::microseconds>(end - start).count();
	}

	node = temp;

	double average_timing = (double)timing / 5;

	cout << "took an average of " << average_timing / 1000 << " milliseconds." << endl;
	cout << "SET_TIMINGS 1 " << depth << " " << width << " " << average_timing / 1000 << endl;

	if (print) {
		cout << endl << "After: " << endl;
		print_cpu_tree(node, 1);
	}
}

struct GPUNode {
	int depth;
	int width;
	long long count;
	thrust::device_vector<int> depths;
	thrust::device_vector<short> types;
	thrust::device_vector<long long> coords;
	GPUNode(int depth, int width);
};

GPUNode::GPUNode(int depth, int width)
	: depth(depth), width(width), count(0)
{
	for (int i = 0; i < depth; i++)
		count += (long long)pow(width, i);

	thrust::host_vector<int> host_depths(count);
	thrust::host_vector<short> host_types(count);
	thrust::host_vector<long long> host_coords(count * depth, 0);

	vector<int> cur_width(depth, 0);
	vector<long long> cur_coord(depth, 0);
	int cur_depth = 0;

	for (int i = 0; i < count; i++) {
		if (cur_width[cur_depth] >= width) {
			cur_coord[cur_depth - 1] += cur_coord[cur_depth];
			cur_coord[cur_depth] = 0;
			cur_depth--;
			i--;
			continue;
		}

		host_depths[i] = cur_depth;
		host_types[i] = cur_depth + 1 >= depth ? 1 : 0;
		cur_coord[cur_depth]++;
		cur_width[cur_depth]++;

		host_coords[i*depth + cur_depth] = cur_coord[0];
		for (int j = cur_depth - 1; j >= 0; j--)
			host_coords[i*depth + j] = host_coords[i*depth + j + 1] + cur_coord[cur_depth - j];

		if (cur_depth + 1 < depth) {
			cur_width[++cur_depth] = 0;
		}
	}

	depths = host_depths;
	types = host_types;
	coords = host_coords;
}

struct print_gpu_node {
	int max_depth;
	thrust::host_vector<long long>& coords;

	template <typename Tuple>
	__host__
	void operator()(Tuple t) 
	{
		int depth = thrust::get<0>(t) + 1;

		for (int i = 0; i < depth; i++)
			cout << " ";

		cout << thrust::get<1>(t) << " ";

		long long i = thrust::get<2>(t);

		for (int j = 0; j < max_depth; j++) {
			long long c = coords[i*max_depth + j];
			if (c)
				cout << " " << c;
			else
				break;
		}

		cout << endl;
	}
};

void print_gpu_tree(GPUNode& ast)
{
	thrust::host_vector<int> host_depths = ast.depths;
	thrust::host_vector<short> host_types = ast.types;
	thrust::host_vector<long long> host_coords = ast.coords;
	thrust::counting_iterator<long long> row(0);

	thrust::for_each(thrust::host,
		thrust::make_zip_iterator(
			thrust::make_tuple(host_depths.begin(), host_types.begin(), row)),
		thrust::make_zip_iterator(
			thrust::make_tuple(host_depths.end(), host_types.end(), row + ast.count)),
		print_gpu_node{ ast.depth, host_coords});
}

struct coord_parent_index : public thrust::unary_function<long long, long long> {
	thrust::device_ptr<long long> coords;
	thrust::device_ptr<int> depths;
	thrust::device_ptr<short> types;
	int max_depth;

	coord_parent_index(
		thrust::device_ptr<long long> cs, 
		thrust::device_ptr<int> depths, 
		thrust::device_ptr<short> types, 
		int md)
		: max_depth(md), depths(depths), types(types), coords(cs)
	{}

	__host__ __device__
	long long operator()(long long i) const
	{
		for (int j = 1; j <= depths[i]; j++) {
			auto parent = coords[i * max_depth + j] - 1;
			if (!types[parent])
				return parent;
		}

		return 0;
	}
};

struct copy_coord {
	thrust::device_ptr<long long> new_coords;
	thrust::device_ptr<long long> old_coords;
	int max_depth;

	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple t)
	{
		long long ci = thrust::get<0>(t);
		long long ei = thrust::get<1>(t);
		for (int j = 0; j < max_depth; j++)
			new_coords[ci*max_depth + j] = old_coords[ei*max_depth + j];
	}
};

void gpu_flatten(GPUNode& ast)
{
	thrust::device_vector<long long> eids(ast.count);
	thrust::sequence(eids.begin(), eids.end());

	auto eids_begin = eids.begin();
	auto eids_end = thrust::remove_if(eids_begin, eids.end(), ast.types.begin(), 
		thrust::identity<short>());

	long long exp_count = eids_end - eids_begin;
	long long result_count = ast.count + exp_count - 1;

	thrust::device_vector<int> new_depths(result_count);
	thrust::device_vector<short> new_types(result_count);
	thrust::device_vector<long long> new_coords(result_count * ast.depth);
	thrust::device_vector<long long> refids(result_count);
	thrust::counting_iterator<long long> newids(0);
	thrust::device_vector<long long> keys(result_count);

	thrust::fill(new_depths.begin(), new_depths.begin() + exp_count, 0);
	thrust::fill(new_depths.begin() + exp_count, new_depths.end(), 1);
	thrust::fill(new_types.begin(), new_types.begin() + exp_count, 0);
	thrust::fill(new_types.begin() + exp_count, new_types.end(), 1);

	auto keys_first = keys.begin() + exp_count;

	thrust::copy(eids_begin, eids_end, keys.begin());
	thrust::transform(newids + 1, newids + ast.count, keys_first,
		coord_parent_index(ast.coords.data(), ast.depths.data(), ast.types.data(), ast.depth));
	thrust::copy(eids_begin, eids_end, refids.begin());
	thrust::sequence(refids.begin() + exp_count, refids.end(), 1);
	thrust::stable_sort_by_key(keys.begin(), keys.end(),
		thrust::make_zip_iterator(
			thrust::make_tuple(new_depths.begin(), new_types.begin(), refids.begin())));
	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(newids, refids.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(newids + result_count, refids.end())),
		copy_coord{ new_coords.data(), ast.coords.data(), ast.depth });

	ast.count = result_count;
	ast.types = new_types;
	ast.coords = new_coords;
	ast.depths = new_depths;
}

void benchmark_gpu(int depth, int width, bool print)
{
	hipSetDevice(1);

	cout << "Benchmarking GPU algorithm (Depth: " << depth << " Width: " << width << ")..." << endl;
	cout << "Creating AST ";

	GPUNode ast(depth+1, width);

	cout << "done." << endl;

	if (print) {
		cout << endl << "Before: " << endl;
		print_gpu_tree(ast);
		cout << endl;
	}

	cout << "Flattening AST...";

	long long timing = 0;

	GPUNode temp(1, 1);

	for (int i = 0; i < 5; i++) {
		GPUNode temp = ast;
		auto start = chrono::high_resolution_clock::now();
		gpu_flatten(temp);
		auto end = chrono::high_resolution_clock::now();
		timing += chrono::duration_cast<chrono::microseconds>(end - start).count();
	}

	double average_timing = (double)timing / 5;

	cout << "took an average of " << average_timing / 1000 << " milliseconds." << endl;
	cout << "SET_TIMINGS 2 " << depth << " " << width << " " << average_timing / 1000 << endl;

	if (print) {
		gpu_flatten(ast);
		cout << endl << "After: " << endl;
		print_gpu_tree(ast);
	}
}

void print_count(int depth, int width)
{
	long long count = 0;

	for (int i = 0; i <= depth; i++)
		count += (long long)pow(width, i);

	cout << "ASTs should have " << count << " elements." << endl << endl;

}

int main(int argc, char *argv[])
{
	int depth = 3;
	int width = 2;
	int depth_end = 4;
	int width_end = 3;
	bool print = true;

	string print_str("print");
	string quiet_str("quiet");
	string bench_str("bench");
	string fnc_str("fnc");
	string mut_str("mut");
	string gpu_str("gpu");

	if (argc != 3 && argc != 5 && argc != 7) {
		help();
		return 1;
	}

	if (argc == 5) {
		depth = stoi(argv[3]);
		width = stoi(argv[4]);
		depth_end = depth + 1;
		width_end = width + 1;
	}

	if (argc == 7) {
		depth = stoi(argv[3]);
		depth_end = stoi(argv[4]);
		width = stoi(argv[5]);
		width_end = stoi(argv[6]);
	}

	if (print_str == argv[2])
		print = true;
	else if (quiet_str == argv[2])
		print = false;
	else {
		help();
		return 1;
	}

	if (fnc_str == argv[1]) {
		print_count(depth, width);
		for (int i = depth; i < depth_end; i++) {
			for (int j = width; j < width_end; j++) {
				benchmark_functional(i, j, print);
			}
		}
	}
	else if (mut_str == argv[1]) {
		print_count(depth, width);
		for (int i = depth; i < depth_end; i++) {
			for (int j = width; j < width_end; j++) {
				benchmark_mutation(i, j, print);
			}
		}
	}
	else if (gpu_str == argv[1]) {
		print_count(depth, width);
		for (int i = depth; i < depth_end; i++) {
			for (int j = width; j < width_end; j++) {
				benchmark_gpu(i, j, print);
			}
		}
	}
	else if (bench_str == argv[1]) {
		print_count(depth, width);
		for (int i = depth; i < depth_end; i++) {
			for (int j = width; j < width_end; j++) {
				benchmark_functional(i, j, print);
				cout << endl;
				benchmark_mutation(i, j, print);
				cout << endl;
				benchmark_gpu(i, j, print);
			}
		}
	}
	else {
		help();
		return 1;
	}

	return 0;
}